#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright 2010 Duane Merrill
 * 
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License. 
 * 
 * 
 * 
 * 
 * AUTHORS' REQUEST: 
 * 
 * 		If you use|reference|benchmark this code, please cite our Technical 
 * 		Report (http://www.cs.virginia.edu/~dgm4d/papers/RadixSortTR.pdf):
 * 
 *		@TechReport{ Merrill:Sorting:2010,
 *        	author = "Duane Merrill and Andrew Grimshaw",
 *        	title = "Revisiting Sorting for GPGPU Stream Architectures",
 *        	year = "2010",
 *        	institution = "University of Virginia, Department of Computer Science",
 *        	address = "Charlottesville, VA, USA",
 *        	number = "CS2010-03"
 *		}
 * 
 * For more information, see our Google Code project site: 
 * http://code.google.com/p/back40computing/
 * 
 * Thanks!
 ******************************************************************************/



/******************************************************************************
 * Radix Sorting API
 *
 ******************************************************************************/

#pragma once

#include <stdlib.h> 
#include <stdio.h> 
#include <string.h> 
#include <math.h> 
#include <float.h>

#include <b40c/KernelCommon/b40c_error_synchronize.cu>
#include <b40c/LsbRadixSort/radixsort_multi_cta.cu>
#include <b40c/LsbRadixSort/kernel/radixsort_singlegrid_kernel.cu>

namespace b40c {

/**
 * Single-grid sorting enactor class.  
 * 
 * This sorting implementation is specifically designed for small problems 
 * that are not large enough to saturate the GPU (e.g., problems < 1M elements.)
 * It performs multiple digit-place passes over the input problem all within
 * a single kernel launch.  It does so by implementing software global-barriers
 * across threadblocks.  
 * 
 * NOTE: This enactor can NOT be used to sort:
 * 		- Problems having structured value-types (i.e., keys that are paired with 
 * 			satellite values that are structs or classes).  This is because
 * 			the compiler cannot be told how to copy structures from global 
 * 			memory using volatile or cache-global load modifiers. 
 * 
 * It also allows the caller to specify the lower-order bits over which the 
 * keys should be sorted (e.g., the lower 17 bits out of 32-bit keys).  This
 * reduces the number of overall sorting passes (4-bits per pass) for 
 * scenarios in which the keyspace can be restricted in this manner.  
 * 
 * To use, simply create a specialized instance of this class with your 
 * key-type K (and optionally value-type V if sorting with satellite 
 * values).  E.g., for sorting signed ints:
 * 
 * 		SingleGridRadixSortingEnactor<int> sorting_enactor;
 * 
 * or for sorting floats paired with unsigned ints:
 * 			
 * 		SingleGridRadixSortingEnactor<float, unsigned int> sorting_enactor;
 * 
 * The enactor itself manages a small amount of device state for use when 
 * performing sorting operations.  To minimize GPU allocation overhead, 
 * enactors can be re-used over multiple sorting operations.  
 * 
 * The problem-storage for a sorting operation is independent of the sorting
 * enactor.  A single enactor can be reused to sort multiple instances of the
 * same type of problem storage.  The MultiCtaRadixSortStorage structure
 * is used to manage the input/output/temporary buffers needed to sort 
 * a problem of a given size.  This enactor will lazily allocate any NULL
 * buffers contained within a problem-storage structure.  
 *
 * Sorting is invoked upon a problem-storage as follows:
 * 
 * 		sorting_enactor.EnactSort(device_storage);
 * 
 * or
 * 
 * 		sorting_enactor.EnactSort<17>(device_storage);
 * 
 * in the case where the caller knows that only the lower 17 bits
 * are used to differentiate keys.  N.B.: for use within templated
 * functions, the proper syntax is:
 *  
 * 		sorting_enactor.template EnactSort<17>(device_storage);
 * 
 * This enactor will update the selector within the problem storage
 * to indicate which buffer contains the sorted output. E.g., 
 * 
 * 		device_storage.d_keys[device_storage.selector];
 * 
 * Please see the overview of MultiCtaRadixSortStorage for more details.
 * 
 * 
 * @template-param K
 * 		Type of keys to be sorted
 * @template-param V
 * 		Type of values to be sorted.
 * @template-param ConvertedKeyType
 * 		Leave as default to effect necessary enactor specialization for 
 * 		signed and floating-point types
 */
template <typename K, typename V = KeysOnlyType>
class SingleGridRadixSortingEnactor;



/**
 * Template-specialized structure for invoking the single-grid kernel a 
 * specific number of times.  We extract this so as to avoid unnecessary
 * kernel generation.
 */
template<int INVOCATIONS, typename K, typename V, int RADIX_BITS, int PASSES> 
struct SingleGridKernelInvoker;


/**
 * Single-grid sorting enactor class.  
 */
template <typename K, typename V>
class SingleGridRadixSortingEnactor : public MultiCtaRadixSortingEnactor<K, V>
{
private:
	
	// Typedef for base class
	typedef MultiCtaRadixSortingEnactor<K, V> Base; 

protected:
	
	// Array of global synchronization counters, one for each threadblock
	int *d_sync;

	
public: 
	
	// Unsigned integer type suitable for radix sorting of keys
	typedef typename KeyConversion<K>::UnsignedBits ConvertedKeyType;

	
	/**
	 * Utility function: Returns the maximum problem size this enactor can sort on the device
	 * it was initialized for.
	 */
	static long long MaxProblemSize(const CudaProperties &props) 
	{
		long long element_size = (Base::KeysOnly()) ? sizeof(K) : sizeof(K) + sizeof(V);

		// Begin with device memory, subtract 128MB for video/spine/etc.  Factor in 
		// three vectors for keys (and values, if present)
		long long available_bytes = props.device_props.totalGlobalMem - 128;
		return available_bytes / (element_size * 3);
	}


protected:
	
	// Radix bits per pass
	static const int RADIX_BITS = 4;
	
	
	/**
	 * Utility function: Returns the default maximum number of threadblocks 
	 * this enactor class can launch.
	 */
	static int MaxGridSize(const CudaProperties &props, int max_grid_size = 0) 
	{
		if (max_grid_size == 0) {
			// No override: Fully populate all SMs
			max_grid_size = props.device_props.multiProcessorCount * 
					B40C_RADIXSORT_SG_OCCUPANCY(props.kernel_ptx_version); 
		} 
		return max_grid_size;
	}
	
	
protected:
	
	/**
	 * Determines the actual number of CTAs to launch for the given problem size
	 * 
	 * @return The actual number of CTAs that should be launched
	 */
	int GridSize(int num_elements)
	{
		// Initially assume that each threadblock will do only one 
		// tile worth of work (and that the last one will do any remainder), 
		// but then clamp it by the "max" restriction  

		int grid_size = (num_elements + this->tile_elements - 1) / this->tile_elements;
		
		if (grid_size == 0) {
			// Always at least one block to process the remainder
			grid_size = 1;
		} else if (grid_size > this->max_grid_size) {
			grid_size = this->max_grid_size;
		}
		
		return grid_size;
	}

	
	/**
     * Post-sorting logic.
     */
    virtual hipError_t PostSort(MultiCtaRadixSortStorage<K, V> &problem_storage, int passes) 
    {
		problem_storage.selector = passes & 0x1;
		return Base::PostSort(problem_storage, passes);
    }

public:

	/**
	 * Constructor.
	 */
	SingleGridRadixSortingEnactor(
		int max_grid_size = 0,
		const CudaProperties &props = CudaProperties()) :
			Base::MultiCtaRadixSortingEnactor(
				MaxGridSize(props, max_grid_size),
				B40C_RADIXSORT_SG_TILE_ELEMENTS(props.kernel_ptx_version , ConvertedKeyType, V),
				RADIX_BITS,
				props), 
			d_sync(NULL)
	{
		// Allocate and initialize synchronization counters
		hipMalloc((void**) &d_sync, sizeof(int) * this->max_grid_size);
		InitSync<void><<<this->max_grid_size, 32, 0>>>(d_sync);
	}

	
	/**
     * Destructor
     */
    virtual ~SingleGridRadixSortingEnactor() 
    {
    	if (d_sync) hipFree(d_sync);
    }
    
	
	/**
	 * Enacts a radix sorting operation on the specified device data.
	 *
	 * @return hipSuccess on success, error enumeration otherwise
	 */
	template <int LOWER_KEY_BITS>
	hipError_t EnactSort(MultiCtaRadixSortStorage<K, V> &problem_storage) 
	{
		const int PASSES = (LOWER_KEY_BITS + RADIX_BITS - 1) / RADIX_BITS;

		// Compute work distribution
		CtaDecomposition work_decomposition;
		int grid_size = GridSize(problem_storage.num_elements);	
		GetWorkDecomposition(problem_storage.num_elements, grid_size, work_decomposition);
		
		// Compute number of spine elements to scan during this pass
		int spine_elements = grid_size * (1 << RADIX_BITS);
		int spine_tiles = (spine_elements + B40C_RADIXSORT_SPINE_TILE_ELEMENTS - 1) / 
				B40C_RADIXSORT_SPINE_TILE_ELEMENTS;
		spine_elements = spine_tiles * B40C_RADIXSORT_SPINE_TILE_ELEMENTS;

		// Perform any lazy allocation
		PreSort(problem_storage, PASSES);
		
		if (RADIXSORT_DEBUG) {
    		printf("\ndevice_sm_version: %d, kernel_ptx_version: %d\n", 
    			this->cuda_props.device_sm_version, this->cuda_props.kernel_ptx_version);
    		printf("%d-bit bottom-level reduction & scan kernels:\n\tgrid_size: %d, \n\tthreads: %d, \n\ttile_elements: %d, \n\tnum_big_blocks: %d, \n\tbig_block_elements: %d, \n\tnormal_block_elements: %d\n\textra_elements_last_block: %d\n\n",
    			LOWER_KEY_BITS, grid_size, B40C_RADIXSORT_THREADS, this->tile_elements, work_decomposition.num_big_blocks, work_decomposition.big_block_elements, work_decomposition.normal_block_elements, work_decomposition.extra_elements_last_block);
    		printf("Top-level spine scan:\n\tgrid_size: %d, \n\tthreads: %d, \n\tspine_block_elements: %d\n\n", 
    			grid_size, B40C_RADIXSORT_SPINE_THREADS, spine_elements);
    	}	
		
		// Invoke kernel
		SingleGridKernelInvoker<(PASSES + 8 - 1) / 8, K, V, RADIX_BITS, PASSES>::Invoke(
			grid_size,	
			this->d_sync,
			this->d_spine, 
			problem_storage, 
			work_decomposition, 
			spine_elements);

	    // Perform any post-mortem
		PostSort(problem_storage, PASSES);

		return hipSuccess;
	}
	
	
	/**
	 * Enacts a radix sorting operation on the specified device data.
	 *
	 * @return hipSuccess on success, error enumeration otherwise
	 */
	hipError_t EnactSort(MultiCtaRadixSortStorage<K, V> &problem_storage) 
	{
		return EnactSort<sizeof(K) * 8>(problem_storage);	// mooch
	}
	
};



/**
 * Template specialization for one invocation of the sorting kernel (which 
 * performs up to 8 passes).
 */
template<typename K, typename V, int RADIX_BITS, int PASSES> 
struct SingleGridKernelInvoker <1, K, V, RADIX_BITS, PASSES>
{
	typedef typename KeyConversion<K>::UnsignedBits ConvertedKeyType;

	static void Invoke(
		int grid_size,	
		int *d_sync,
		int *d_spine, 
		MultiCtaRadixSortStorage<K, V> &problem_storage, 
		CtaDecomposition &work_decomposition, 
		int spine_elements)
	{
		LsbSingleGridSortingKernel<ConvertedKeyType, V, RADIX_BITS, PASSES, 0, PreprocessKeyFunctor<K>, PostprocessKeyFunctor<K> ><<<grid_size, B40C_RADIXSORT_THREADS, 0>>>(
			d_sync,
			d_spine,
			(ConvertedKeyType *) problem_storage.d_keys[0],
			(ConvertedKeyType *) problem_storage.d_keys[1],
			problem_storage.d_values[0],
			problem_storage.d_values[1],
			work_decomposition,
			spine_elements);
	    synchronize_if_enabled("ScanScatterDigits");
	}
};


/**
 * Template specialization for two invocations of the sorting kernel (which 
 * performs up to 8 passes).
 */
template<typename K, typename V, int RADIX_BITS, int PASSES> 
struct SingleGridKernelInvoker <2, K, V, RADIX_BITS, PASSES>
{
	typedef typename KeyConversion<K>::UnsignedBits ConvertedKeyType;

	static void Invoke(
		int grid_size,	
		int *d_sync,
		int *d_spine, 
		MultiCtaRadixSortStorage<K, V> &problem_storage, 
		CtaDecomposition &work_decomposition, 
		int spine_elements)
	{
		LsbSingleGridSortingKernel<ConvertedKeyType, V, RADIX_BITS, PASSES, 0, PreprocessKeyFunctor<K>, PostprocessKeyFunctor<K> ><<<grid_size, B40C_RADIXSORT_THREADS, 0>>>(
			d_sync,
			d_spine,
			(ConvertedKeyType *) problem_storage.d_keys[0],
			(ConvertedKeyType *) problem_storage.d_keys[1],
			problem_storage.d_values[0],
			problem_storage.d_values[1],
			work_decomposition,
			spine_elements);
	    synchronize_if_enabled("ScanScatterDigits");

	    LsbSingleGridSortingKernel<ConvertedKeyType, V, RADIX_BITS, PASSES, 8, PreprocessKeyFunctor<K>, PostprocessKeyFunctor<K> ><<<grid_size, B40C_RADIXSORT_THREADS, 0>>>(
			d_sync,
			d_spine,
			(ConvertedKeyType *) problem_storage.d_keys[0],
			(ConvertedKeyType *) problem_storage.d_keys[1],
			problem_storage.d_values[0],
			problem_storage.d_values[1],
			work_decomposition,
			spine_elements);
	    synchronize_if_enabled("ScanScatterDigits");
	}
};




}// namespace b40c

