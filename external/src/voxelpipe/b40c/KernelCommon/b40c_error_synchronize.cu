/**
 * Copyright 2010 Duane Merrill
 * 
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License. 
 * 
 * For more information, see our Google Code project site: 
 * http://code.google.com/p/back40computing/
 * 
 * Thanks!
 */

#pragma once

namespace b40c {

inline void synchronize(const char *message)
{
	hipError_t error = hipDeviceSynchronize();
	if(error) {
		fprintf(stderr, "%s caused %d (%s)\n", message, error, hipGetErrorString(error));
//		exit(1);
	}
} 

inline void synchronize_if_enabled(const char *message)
{
	// XXX this could potentially be a runtime decision
#if defined(__ERROR_SYNCHRONOUS)
	synchronize(message);
#endif
} 



} // namespace b40c

