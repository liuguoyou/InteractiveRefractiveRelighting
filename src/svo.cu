#include "hip/hip_runtime.h"

#include "svo.h"
#include "voxelization.h"

#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>

#include <stack>

#include "timingUtils.h"

__global__ void flagNodes(int* voxels, int numVoxels, int* octree, int M, int T, float3 bbox0, float3 t_d, float3 p_d, int tree_depth) {

  int index = blockIdx.x * blockDim.x + threadIdx.x;

  //Don't do anything if its out of bounds
  if (index < numVoxels) {
    float3 center = getCenterFromIndex(voxels[index], M, T, bbox0, t_d, p_d);
    float edge_length = abs(bbox0.x);
    float3 center_depth = make_float3(0.0f, 0.0f, 0.0f);
    int node_idx = 0;
    int this_node;

    //Loop until the specified depth
    for (int i = 0; i < tree_depth; i++) {
      int x = center.x > center_depth.x;
      int y = center.y > center_depth.y;
      int z = center.z > center_depth.z;
      this_node = node_idx + (x + (y << 1) + (z << 2));

      if (i < tree_depth - 1) {
        //The lowest 30 bits are the address
        node_idx = octree[2 * this_node] & 0x3FFFFFFF;

        //Update the center depth for the next iteration
        center_depth.x += edge_length / 2 * (x ? 1 : -1);
        center_depth.y += edge_length / 2 * (y ? 1 : -1);
        center_depth.z += edge_length / 2 * (z ? 1 : -1);
      }
      edge_length /= 2.0f;
    }
    octree[2 * this_node] = octree[2 * this_node] | 0x40000000;
  }

}

__global__ void splitNodes(int* octree, int* numNodes, int poolSize, int startNode) {

  int index = blockIdx.x * blockDim.x + threadIdx.x;

  //Don't do anything if its out of bounds
  if (index < poolSize) {
    int node = octree[2 * (index+startNode)];

    //Split the node if its flagged
    if (node & 0x40000000) {
      //Get a new node tile
      int newNode = atomicAdd(numNodes, 8);

      //Point this node at the new tile
      octree[2 * (index+startNode)] = (octree[2 * (index+startNode)] & 0xC0000000) | (newNode & 0x3FFFFFFF);

      //Initialize new child nodes to 0's
      for (int off = 0; off < 8; off++) {
        octree[2 * (newNode + off)] = 0;
        octree[2 * (newNode + off) + 1] = 0;
      }
    }
  }

}

__global__ void fillNodes(int* voxels, int numVoxels, int* values, int* octree, int M, int T, float3 bbox0, float3 t_d, float3 p_d) {

  int index = blockIdx.x * blockDim.x + threadIdx.x;

  //Don't do anything if its out of bounds
  if (index < numVoxels) {
    float3 center = getCenterFromIndex(voxels[index], M, T, bbox0, t_d, p_d);
    float edge_length = abs(bbox0.x);
    float3 center_depth = make_float3(0.0f, 0.0f, 0.0f);
    int node_idx = 0;
    int this_node;
    bool has_child = true;

    //Loop until the specified depth
    while (has_child) {
      int x = center.x > center_depth.x;
      int y = center.y > center_depth.y;
      int z = center.z > center_depth.z;
      this_node = node_idx + (x + (y << 1) + (z << 2));

      has_child = octree[2 * this_node] & 0x40000000;

      if (has_child) {
        //The lowest 30 bits are the address
        node_idx = octree[2 * this_node] & 0x3FFFFFFF;

        //Update the center depth for the next iteration
        center_depth.x += edge_length / 2 * (x ? 1 : -1);
        center_depth.y += edge_length / 2 * (y ? 1 : -1);
        center_depth.z += edge_length / 2 * (z ? 1 : -1);
      }
      edge_length /= 2.0f;
    }
    octree[2 * this_node + 1] = values[index];
  }

}

__global__ void mipmapNodes(int* octree, int poolSize, int startNode) {

  int index = blockIdx.x * blockDim.x + threadIdx.x;

  //Don't do anything if its out of bounds
  if (index < poolSize) {
    int node = octree[2 * (index + startNode)];

    //Don't do anything if this node has no children
    if (!(node & 0x40000000)) {
      return;
    }

    //Get the child pointer
    int childPoint = (node & 0x3FFFFFFF);

    //Loop through children values and average them
    glm::vec4 val = glm::vec4(0.0);
    for (int i = 0; i < 8; i++) {
      int child_val = octree[2*(childPoint+i) + 1];
      val.r += (float) (child_val & 0xFF) / 8.0f;
      val.g += (float) ((child_val >> 8) & 0xFF) / 8.0f;
      val.b += (float) ((child_val >> 16) & 0xFF) / 8.0f;
      val.a += (float) ((child_val >> 24) & 0x7F) / 8.0f;
    }

    //Assign value of this node to the average
    int r = (int) (val.r);
    int g = (int) (val.g);
    int b = (int) (val.b);
    int a = (int) (val.a);
    octree[(2 * (index + startNode)) + 1] = r + (g << 8) + (b << 16) + (a << 24);
  }

}

__global__ void createCubeMeshFromSVO(int* octree, int* counter, int total_depth, float3 bbox0, float cube_scale, int num_voxels, float* cube_vbo,
  int cube_vbosize, int* cube_ibo, int cube_ibosize, float* cube_nbo, float* out_vbo, int* out_ibo, float* out_nbo, float* out_cbo) {

  //Get the index for the thread
  int idx = blockIdx.x*blockDim.x + threadIdx.x;

  float edge_length = abs(bbox0.x);
  float3 center = make_float3(0.0f, 0.0f, 0.0f);
  int pointer = 0;
  bool has_child = true;
  int val2;
  int depth = 0;

  while (has_child && depth++ < total_depth) {
    //Get the lowest 3 bits to encode the first move
    int pos = idx & 0x7;

    //Get the value from the octree
    int val = octree[2*(pointer+pos)];
    val2 = octree[2*(pointer + pos) + 1];

    //It it is not occupied, do not continue
    has_child = val & 0x40000000;

    //Don't continue if it does not have a child
    if (!has_child && (octree[2 * (pointer + pos) + 1] >> 24 == 0)) {
      return;
    }

    //Get the child pointer for the next depth
    pointer = val & 0x3FFFFFFF;

    //Decode the value into xyz
    int x = pos & 0x1;
    int y = pos & 0x2;
    int z = pos & 0x4;

    //Update the center
    center.x += edge_length / 2 * (x ? 1 : -1);
    center.y += edge_length / 2 * (y ? 1 : -1);
    center.z += edge_length / 2 * (z ? 1 : -1);

    //Half the edge length for the next iteration
    edge_length /= 2.0f;

    //Shift right for the next iteration
    idx = idx >> 3;
  }

  int vidx = atomicAdd(counter, 1);
  //TODO: Detect if we exceed the allocated memory, and break out + allocate more

  if (vidx < num_voxels) {

    int vbo_offset = vidx * cube_vbosize;
    int ibo_offset = vidx * cube_ibosize;

    for (int i = 0; i < cube_vbosize; i++) {
      if (i % 3 == 0) {
        out_vbo[vbo_offset + i] = cube_vbo[i] * (edge_length/cube_scale) + center.x;
        out_cbo[vbo_offset + i] = (float)((val2 & 0xFF) / 255.0);
      }
      else if (i % 3 == 1) {
        out_vbo[vbo_offset + i] = cube_vbo[i] * (edge_length/cube_scale) + center.y;
        out_cbo[vbo_offset + i] = (float)(((val2 >> 8) & 0xFF) / 255.0);
      }
      else {
        out_vbo[vbo_offset + i] = cube_vbo[i] *(edge_length/cube_scale) + center.z;
        out_cbo[vbo_offset + i] = (float)(((val2 >> 16) & 0xFF) / 255.0);
      }
      out_nbo[vbo_offset + i] = cube_nbo[i];
    }

    for (int i = 0; i < cube_ibosize; i++) {
      out_ibo[ibo_offset + i] = cube_ibo[i] + ibo_offset;
    }

  }

}

//This is based on Cyril Crassin's approach
__host__ void svoFromVoxels(int* d_voxels, int numVoxels, int* d_values, int* d_octree) {
  int numNodes = 8;
  std::stack<int> startingNodes;
  startingNodes.push(0);
  int* d_numNodes;
  hipMalloc((void**)&d_numNodes, sizeof(int));
  hipMemcpy(d_numNodes, &numNodes, sizeof(int), hipMemcpyHostToDevice);
  int depth = 0;

  while (numNodes < (numVoxels*log_N) && ++depth < log_N) {

    //First, parallelize on voxels and flag nodes to be subdivided
    flagNodes<<<(numVoxels / 256) + 1, 256>>>(d_voxels, numVoxels, d_octree, M, T, bbox0, t_d, p_d, depth);

    hipDeviceSynchronize();

    //Then, parallize on nodes and subdivide
    splitNodes<<<((numNodes - startingNodes.top()) / 256) + 1, 256>>>(d_octree, d_numNodes, numNodes - startingNodes.top(), startingNodes.top());
    startingNodes.push(numNodes);

    hipDeviceSynchronize();
    hipMemcpy(&numNodes, d_numNodes, sizeof(int), hipMemcpyDeviceToHost);
  }

  std::cout << "Num Nodes: " << numNodes << std::endl;

  //Write voxel values into the lowest level of the svo
  fillNodes<<<(numVoxels / 256) + 1, 256>>>(d_voxels, numVoxels, d_values, d_octree, M, T, bbox0, t_d, p_d);
  hipDeviceSynchronize();

  //Loop through the levels of the svo bottom to top and map the values by averaging child values
  numNodes = startingNodes.top(); //Skip the lowest level since the fillNodes() kernel handled that level
  startingNodes.pop();
  while (!startingNodes.empty()) {
    mipmapNodes << <((numNodes - startingNodes.top()) / 256) + 1, 256 >> >(d_octree, numNodes - startingNodes.top(), startingNodes.top());
    hipDeviceSynchronize();

    numNodes = startingNodes.top();
    startingNodes.pop();
  }

  hipFree(d_numNodes);
}

__host__ void extractCubesFromSVO(int* d_octree, int numVoxels, Mesh &m_cube, Mesh &m_out) {

  //Move cube data to GPU
  thrust::device_vector<float> d_vbo_cube(m_cube.vbo, m_cube.vbo + m_cube.vbosize);
  thrust::device_vector<int> d_ibo_cube(m_cube.ibo, m_cube.ibo + m_cube.ibosize);
  thrust::device_vector<float> d_nbo_cube(m_cube.nbo, m_cube.nbo + m_cube.nbosize);

  //Create output structs
  float* d_vbo_out;
  int* d_ibo_out;
  float* d_nbo_out;
  float* d_cbo_out;
  hipMalloc((void**)&d_vbo_out, numVoxels * m_cube.vbosize * sizeof(float));
  hipMalloc((void**)&d_ibo_out, numVoxels * m_cube.ibosize * sizeof(int));
  hipMalloc((void**)&d_nbo_out, numVoxels * m_cube.nbosize * sizeof(float));
  hipMalloc((void**)&d_cbo_out, numVoxels * m_cube.nbosize * sizeof(float));

  //Warn if vbo and nbo are not same size on cube
  if (m_cube.vbosize != m_cube.nbosize) {
    std::cout << "ERROR: cube vbo and nbo have different sizes." << std::endl;
    return;
  }

  //Create global counter to determine where to write the output
  int* d_counter;
  int initial_count = 0;
  hipMalloc((void**)&d_counter, sizeof(int));
  hipMemcpy(d_counter, &initial_count, sizeof(int), hipMemcpyHostToDevice);

  //Determine how to scale the number of threads needed based on the octree depth to render
  int fac = (log_N > log_SVO_N) ? pow(8, log_N - log_SVO_N) : 1;

  //Create resulting cube-ized mesh
  createCubeMeshFromSVO << <(N*N*N / 256 / fac) + 1, 256 >> >(d_octree, d_counter, log_SVO_N, bbox0, CUBE_MESH_SCALE, numVoxels, thrust::raw_pointer_cast(&d_vbo_cube.front()),
    m_cube.vbosize, thrust::raw_pointer_cast(&d_ibo_cube.front()), m_cube.ibosize, thrust::raw_pointer_cast(&d_nbo_cube.front()), d_vbo_out, d_ibo_out, d_nbo_out, d_cbo_out);

  //Store output sizes
  m_out.vbosize = numVoxels * m_cube.vbosize;
  m_out.ibosize = numVoxels * m_cube.ibosize;
  m_out.nbosize = numVoxels * m_cube.nbosize;
  m_out.cbosize = m_out.nbosize;

  //Memory allocation for the outputs
  m_out.vbo = (float*)malloc(m_out.vbosize * sizeof(float));
  m_out.ibo = (int*)malloc(m_out.ibosize * sizeof(int));
  m_out.nbo = (float*)malloc(m_out.nbosize * sizeof(float));
  m_out.cbo = (float*)malloc(m_out.cbosize * sizeof(float));

  //Sync here after doing some CPU work
  hipDeviceSynchronize();

  //Copy data back from GPU
  //TODO: Can we avoid this step by making everything run from device-side VBO/IBO/NBO/CBO?
  hipMemcpy(m_out.vbo, d_vbo_out, m_out.vbosize*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(m_out.ibo, d_ibo_out, m_out.ibosize*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(m_out.nbo, d_nbo_out, m_out.nbosize*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(m_out.cbo, d_cbo_out, m_out.cbosize*sizeof(float), hipMemcpyDeviceToHost);

  ///Free GPU memory
  hipFree(d_vbo_out);
  hipFree(d_ibo_out);
  hipFree(d_nbo_out);
  hipFree(d_counter);
}

__host__ void voxelizeSVOCubes(Mesh &m_in, bmp_texture* tex, Mesh &m_cube, Mesh &m_out) {

  //Voxelize the mesh input
  int numVoxels = N*N*N;
  int* d_voxels;
  int* d_values;
  hipMalloc((void**)&d_voxels, numVoxels*sizeof(int));
  hipMalloc((void**)&d_values, numVoxels*sizeof(int));
  numVoxels = voxelizeMesh(m_in, tex, d_voxels, d_values);

  //Create the octree
  int* d_octree = NULL;
  hipMalloc((void**)&d_octree, 8*log_N*numVoxels*sizeof(int));
  startTiming();
  svoFromVoxels(d_voxels, numVoxels, d_values, d_octree);
  std::cout << "Build SVO Time: " << stopTiming() << std::endl;

  //Extract cubes from the leaves of the octree
  startTiming();
  extractCubesFromSVO(d_octree, numVoxels, m_cube, m_out);
  std::cout << "Extract SVO Time: " << stopTiming() << std::endl;

  //Free up GPU memory
  hipFree(d_voxels);
  hipFree(d_octree);

}
